#include "hip/hip_runtime.h"
#include "../include/pddp.cuh"

__global__ void initialize_w_kernel(matrix M, matrix w)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M.rows) {
        double sum = 0;
        for (size_t col = 0; col < M.cols; ++col) {
            sum += M.data[idx(row, M.cols, col)];
        }
        w.data[row] = sum / M.cols;
    }

}

__global__ void initialize_vector_kernel(matrix x, double value)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < x.rows) {
        x.data[index] = value;
    }
}