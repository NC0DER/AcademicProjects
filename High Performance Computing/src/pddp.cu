#include "hip/hip_runtime.h"
#include "../include/pddp.cuh"

__global__ void initialize_vector_kernel(matrix x, double value)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < x.rows) {
        x.data[index] = value;
    }
}