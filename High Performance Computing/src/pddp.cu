#include "hip/hip_runtime.h"
#include "../include/pddp.cuh"

#ifdef SHARED
__global__ void matrix_multiplication_kernel(matrix w, matrix a, matrix b, matrix c, unsigned int tile_size)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    extern __shared__ double buffer[];
    double *a_shared = &buffer[0];
    double *b_shared = &buffer[tile_size * tile_size];

    double sum = 0;

    for (int k = 0; k < (tile_size + a.cols - 1) / tile_size; k++) {
        if (k * tile_size + tx < a.cols && row < a.rows) {
            a_shared[ty * tile_size + tx] = a.data[row * a.cols + k * tile_size + tx];
        }
        else {
            a_shared[ty * tile_size + tx] = 0.0;
        }
        if (k * tile_size + ty < b.rows && col < b.rows) {
            b_shared[ty * tile_size + tx] = b.data[(k * tile_size + ty) * b.cols + col];
        }
        else {
            b_shared[ty * tile_size + tx] = 0.0;
        }
        __syncthreads();
#pragma unroll
        for (int n = 0; n < tile_size; ++n) {
            sum += (a_shared[ty * tile_size + n] - w.data[row]) * b_shared[n * tile_size + tx];
        }
        __syncthreads();
    }
    if (row < c.rows && col < c.cols) {
        c.data[row * c.cols + col] = sum;
    }
}

__global__ void transposed_matrix_multiplication_kernel(matrix w, matrix a, matrix b, matrix c, unsigned int tile_size)
{
    int bx = blockIdx.x; 
    int tx = threadIdx.x;
    int index = bx * blockDim.x + tx;

    extern __shared__ double shared[];
    
    double sum = 0;

    for (int k = 0; k < (tile_size + a.rows - 1) / tile_size; k++) {
        if (k * tile_size + tx < b.rows) {
            shared[tx] = b.data[k * tile_size + tx];
        }
        else {
            shared[tx] = 0.0;
        }
        __syncthreads();
#pragma unroll
        for (int n = 0; n < tile_size; ++n) {
            if (index + (n + tile_size * k) * a.cols < a.size) {
                sum += (a.data[index + (n + tile_size * k) * a.cols] - w.data[n + tile_size * k]) * shared[n];
            }
        }
        __syncthreads();
    }
    if (index < c.rows) {
        c.data[index] = sum;
    }
}
#else
__global__ void matrix_multiplication_kernel(matrix w, matrix a, matrix b, matrix c)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    if (row < c.rows && col < c.cols) {
        double sum = 0;
#pragma unroll
        for (int k = 0; k < a.cols; k++) { //Common Dimension -> m 
            sum += (a.data[row * a.cols + k] - w.data[row]) * b.data[k * b.cols + col];
        }
        c.data[row * c.cols + col] = sum;
    }
}

__global__ void transposed_matrix_multiplication_kernel(matrix w, matrix a, matrix b, matrix c)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    if (row < c.rows && col < c.cols) {
        double sum = 0;
#pragma unroll
        for (int k = 0; k < a.rows; k++) { //Common Dimension -> n 
            sum += (a.data[k * a.cols + row] - w.data[k]) * b.data[k * b.cols + col];
        }
        c.data[row * c.cols + col] = sum;
    }
}
#endif

__global__ void initialize_w_kernel(matrix M, matrix w)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M.rows) {
        double sum = 0;
        for (size_t col = 0; col < M.cols; ++col) {
            sum += M.data[idx(row, M.cols, col)];
        }
        w.data[row] = sum / M.cols;
    }

}

__global__ void initialize_vector_kernel(matrix x, double value)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < x.rows) {
        x.data[index] = value;
    }
}