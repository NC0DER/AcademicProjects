#include <iostream>
#include "../include/wrappers.cuh"

void initialize_vector(matrix x, double value, unsigned int blockSizeX)
{
    hipError_t cudaStatus;
    // Define grid dimensions
    unsigned int numBlocksX = (x.rows - 1) / blockSizeX + 1;
    dim3 dimGrid(numBlocksX, 1, 1);
    dim3 dimBlock(blockSizeX, 1, 1);
    initialize_vector_kernel << <dimGrid, dimBlock >> > (x, value);

    // hipDeviceSynchronize waits for the kernel to finish.
    // While returing any errors that happended during the kernel launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching initialize_vector Kernel!" << std::endl;
    }
}

void initialize_w(matrix M, matrix w, unsigned int blockSizeX)
{
    hipError_t cudaStatus;
    // Define grid dimensions
    dim3 dimBlock;
    dim3 dimGrid;
    dimBlock.x = blockSizeX;
    dimBlock.y = 1;
    dimBlock.z = 1;
    dimGrid.x = (M.rows - 1) / dimBlock.x + 1;
    dimGrid.y = 1;
    dimGrid.z = 1;
    initialize_w_kernel << <dimGrid, dimBlock >> > (M, w);

    // hipDeviceSynchronize waits for the kernel to finish.
    // While returing any errors that happended during the kernel launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching initialize_w Kernel!" << std::endl;
    }
}

float matrix_multiplication(matrix w, matrix M, matrix x_prev, matrix s, unsigned int blockSize)
{
    hipError_t cudaStatus;
    // Events for measuring the execution time
    hipEvent_t start;
    hipEvent_t stop;
    float msec;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Define grid dimensions
    dim3 dimBlock;
    dim3 dimGrid;
    dimBlock.x = blockSize;
    dimBlock.y = blockSize;
    dimBlock.z = 1;
    dimGrid.x = (s.cols - 1) / dimBlock.x + 1;
    dimGrid.y = (s.rows - 1) / dimBlock.y + 1;
    dimGrid.z = 1;

    initialize_vector(s, 0, 16);
    hipEventRecord(start); // Start Event.
#ifdef SHARED  // tileSize = blockSize
    matrix_multiplication_kernel << <dimGrid, dimBlock, 2 * blockSize * blockSize * sizeof(double) >> >(w, M, x_prev, s, blockSize); 
#else
    matrix_multiplication_kernel << <dimGrid, dimBlock, 2 * blockSize * blockSize * sizeof(double) >> >(w, M, x_prev, s);
#endif
    hipEventRecord(stop); // Stop Event.

    hipEventSynchronize(stop);
    hipEventElapsedTime(&msec, start, stop);

    // hipDeviceSynchronize waits for the kernel to finish.
    // While returing any errors that happended during the kernel launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching MatrixVectorMult Kernel!" << std::endl;
    }
    return msec;
}

float transposed_matrix_multiplication(matrix w, matrix M, matrix s, matrix x, unsigned int blockSize)
{
    hipError_t cudaStatus;
    // Events for measuring the execution time
    hipEvent_t start;
    hipEvent_t stop;
    float msec;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Define grid dimensions
    dim3 dimBlock;
    dim3 dimGrid;
    
    dimBlock.x = blockSize;
    dimBlock.y = blockSize;
    dimBlock.z = 1;
    dimGrid.z = 1;
    dimGrid.x = (x.cols - 1) / dimBlock.x + 1;
    dimGrid.y = (x.rows - 1) / dimBlock.y + 1;
    dimGrid.z = 1;

    hipEventRecord(start); // Start Event.
    transposed_matrix_multiplication_kernel << <dimGrid, dimBlock >> >(w, M, s, x);
    hipEventRecord(stop); // Stop Event.

    hipEventSynchronize(stop);
    hipEventElapsedTime(&msec, start, stop);

    // hipDeviceSynchronize waits for the kernel to finish.
    // While returing any errors that happended during the kernel launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching TransposedMatrixVectorMult Kernel!" << std::endl;
    }
    return msec;
}

float Multiply(matrix M, matrix w, matrix x, matrix x_prev, matrix s, unsigned int blockSize)
{
    float msec[1];
    msec[0] = matrix_multiplication(w, M, x_prev, s, blockSize); // s = (M - w * eT) * x_prev
    msec[1] = transposed_matrix_multiplication(w, M, s, x, blockSize); // (M - w * eT)T * s
    return msec[0] + msec[1]; // Return the total elapsed time of 2 multiplication kernels.
}