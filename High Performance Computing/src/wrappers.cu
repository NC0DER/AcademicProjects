#include <iostream>
#include "../include/wrappers.cuh"

void initialize_vector(matrix x, double value, unsigned int blockSizeX)
{
    hipError_t cudaStatus;
    // Define grid dimensions
    unsigned int numBlocksX = (x.rows - 1) / blockSizeX + 1;
    dim3 dimGrid(numBlocksX, 1, 1);
    dim3 dimBlock(blockSizeX, 1, 1);
    initialize_vector_kernel << <dimGrid, dimBlock >> > (x, value);

    // hipDeviceSynchronize waits for the kernel to finish.
    // While returing any errors that happended during the kernel launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching initialize_vector Kernel!" << std::endl;
    }
}

void initialize_w(matrix M, matrix w, unsigned int blockSizeX)
{
    hipError_t cudaStatus;
    // Define grid dimensions
    dim3 dimBlock;
    dim3 dimGrid;
    dimBlock.x = blockSizeX;
    dimBlock.y = 1;
    dimBlock.z = 1;
    dimGrid.x = (M.rows - 1) / dimBlock.x + 1;
    dimGrid.y = 1;
    dimGrid.z = 1;
    initialize_w_kernel << <dimGrid, dimBlock >> > (M, w);

    // hipDeviceSynchronize waits for the kernel to finish.
    // While returing any errors that happended during the kernel launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching initialize_w Kernel!" << std::endl;
    }
}
