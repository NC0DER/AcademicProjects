#include <iostream>
#include "../include/wrappers.cuh"

void initialize_vector(matrix x, double value, unsigned int blockSizeX)
{
    hipError_t cudaStatus;
    // Define grid dimensions
    unsigned int numBlocksX = (x.rows - 1) / blockSizeX + 1;
    dim3 dimGrid(numBlocksX, 1, 1);
    dim3 dimBlock(blockSizeX, 1, 1);
    initialize_vector_kernel << <dimGrid, dimBlock >> > (x, value);

    // hipDeviceSynchronize waits for the kernel to finish.
    // While returing any errors that happended during the kernel launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching initialize_vector Kernel!" << std::endl;
    }
}
