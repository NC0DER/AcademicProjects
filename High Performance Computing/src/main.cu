#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <cstdlib>

#include "hip/hip_runtime.h"
#include ""
#include "../include/matrix.h"
#include "../include/wrappers.cuh"

#define debug 1

int main(int argc, char *argv[]) {
    // Filename given as input from console.
    if (argc != 3) {
        std::cerr << "Syntax: ./<executable> <filename> <blocksize per dimension> \n"
                  << " *Note: Some kernels use the same blocksize for dimension X and Y \n"
                  << " *Note: The resulting blocksize is (e.g for input n) n x n x 1 for each dim (x, y, z) \n \n"
                  << "**Note: Blocksize for each dimension X & Y much larger than 32 \n"
                  << "**Note: is not supported for cards with comp.capability < 6.0" << std::endl;
        return EXIT_FAILURE;
    }
    // Convert the text argument to integer.
    int block_num = 0;
    std::istringstream block_stream(argv[2]);
    block_stream >> block_num;
    
    // Bounds Checking for the blocksize.
    if (block_num < 0) {
        std::cerr << " *Error: blocksize for each dimension should be a positive number" << std::endl;
        return EXIT_FAILURE;
    }
    else if (block_num < 16) {
        std::cerr << " *Error: the number of threads for each dimension exceeds 65535 \n (e.g 1.000.000 / 15 > 65335) \n"
                  << " *Note:  use a blocksize for each dimension >= 16" << std::endl;
        return EXIT_FAILURE;
    }
    else if (block_num > 32) {
        std::cerr << " *Error: the number of blocks for both dimensions exceeds 1024 \n (e.g 33x33 = 1089) \n"
                  <<  " *Note:  use a blocksize for each dimension <= 32" << std::endl;
        return EXIT_FAILURE;
    }

    std::ifstream file(argv[1]);
    // Checking if file stream has succesfully opened.
    if (!file.is_open()) {
        std::cerr << "File doesn't exist, or failed to open!" << std::endl;
        return EXIT_FAILURE;
    }

    // Load the float values on a dynamically expanding vector.
    std::vector<double> text;
    std::string line;
    double value = 0;
    int n = 0, m = 0;
    while (getline(file, line)) {
        std::stringstream stream(line);
        std::string col;
        m = 0;
        while (getline(stream, col, ',')) {
            std::istringstream num_stream(col);
            num_stream >> value;
            text.push_back(value);
            m++;
        }
        n++;
    }
    file.close();

    matrix M(n, m);
    matrix w(n, 1);
    matrix x(m, 1);
    matrix x_prev(m, 1);

    matrix dev_M(n, m);
    matrix dev_w(n, 1);
    matrix dev_x(m, 1);
    matrix dev_x_prev(m, 1);
    matrix dev_s(n, 1);

    M.data = (double *)malloc(M.size * sizeof(double));
    w.data = (double *)malloc(w.size * sizeof(double));
    x.data = (double *)malloc(x.size * sizeof(double));
    x_prev.data = (double *)malloc(x_prev.size * sizeof(double));

    // Load M on the Host system memory.
    for (int row = 0; row < M.rows; ++row) {
        for (int col = 0; col < M.cols; ++col) {
            M.data[idx(row, M.cols, col)] = text[idx(row, M.cols, col)];
        }
    }
    text.clear(); // Empty the vector.

    // Initialize w values to 0
    for (int index = 0; index < n; ++index) {
        w.data[index] = 0.0;
    }

    hipError_t cudaStatus; // Return value of each cuda call.

    // Choose which GPU Device to execute on.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed!";

        // Free all memory both in Device and CPU.
        hipFree(dev_M.data);
        hipFree(dev_w.data);
        hipFree(dev_x.data);
        hipFree(dev_x_prev.data);

        free(M.data);
        free(w.data);
        free(x.data);
        free(x_prev.data);

        return EXIT_FAILURE;
    }

    hipMalloc(&dev_M.data, dev_M.size * sizeof(double));
    hipMalloc(&dev_w.data, dev_w.size * sizeof(double));
    hipMalloc(&dev_x.data, dev_x.size * sizeof(double));
    hipMalloc(&dev_x_prev.data, dev_x_prev.size * sizeof(double));
    hipMalloc(&dev_s.data, dev_s.size * sizeof(double));

    initialize_vector(dev_x, 0, block_num);
    initialize_vector(dev_x_prev, 1, block_num);

    cudaStatus = hipMemcpy(dev_M.data, M.data, dev_M.size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed! 1";

        // Free all memory both in Device and CPU.
        hipFree(dev_M.data);
        hipFree(dev_w.data);
        hipFree(dev_x.data);
        hipFree(dev_x_prev.data);

        free(M.data);
        free(w.data);
        free(x.data);
        free(x_prev.data);

    initialize_w(dev_M, dev_w, block_num);
    cudaStatus = hipMemcpy(w.data, dev_w.data, dev_w.size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed! 2";

        // Free all memory both in Device and CPU.
        hipFree(dev_M.data);
        hipFree(dev_w.data);
        hipFree(dev_x.data);
        hipFree(dev_x_prev.data);

        free(M.data);
        free(w.data);
        free(x.data);
        free(x_prev.data);

        return EXIT_FAILURE;
    }

        return EXIT_FAILURE;
    }
    hipFree(dev_M.data);
    hipFree(dev_w.data);
    hipFree(dev_x.data);
    hipFree(dev_x_prev.data);
    hipFree(dev_s.data);

    free(M.data);
    free(w.data);
    free(x.data);
    free(x_prev.data);
    return EXIT_SUCCESS;
}