#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <cstdlib>

#include "hip/hip_runtime.h"
#include ""
#include "../include/matrix.h"
#include "../include/wrappers.cuh"

#define debug 1

int main(int argc, char *argv[]) {
    // Filename given as input from console.
    if (argc != 3) {
        std::cerr << "Syntax: ./<executable> <filename> <blocksize per dimension> \n"
                  << " *Note: Some kernels use the same blocksize for dimension X and Y \n"
                  << " *Note: The resulting blocksize is (e.g for input n) n x n x 1 for each dim (x, y, z) \n \n"
                  << "**Note: Blocksize for each dimension X & Y much larger than 32 \n"
                  << "**Note: is not supported for cards with comp.capability < 6.0" << std::endl;
        return EXIT_FAILURE;
    }
    // Convert the text argument to integer.
    int block_num = 0;
    std::istringstream block_stream(argv[2]);
    block_stream >> block_num;
    
    // Bounds Checking for the blocksize.
    if (block_num < 0) {
        std::cerr << " *Error: blocksize for each dimension should be a positive number" << std::endl;
        return EXIT_FAILURE;
    }
    else if (block_num < 16) {
        std::cerr << " *Error: the number of threads for each dimension exceeds 65535 \n (e.g 1.000.000 / 15 > 65335) \n"
                  << " *Note:  use a blocksize for each dimension >= 16" << std::endl;
        return EXIT_FAILURE;
    }
    else if (block_num > 32) {
        std::cerr << " *Error: the number of blocks for both dimensions exceeds 1024 \n (e.g 33x33 = 1089) \n"
                  <<  " *Note:  use a blocksize for each dimension <= 32" << std::endl;
        return EXIT_FAILURE;
    }

    std::ifstream file(argv[1]);
    // Checking if file stream has succesfully opened.
    if (!file.is_open()) {
        std::cerr << "File doesn't exist, or failed to open!" << std::endl;
        return EXIT_FAILURE;
    }

    // Load the float values on a dynamically expanding vector.
    std::vector<double> text;
    std::string line;
    double value = 0;
    int n = 0, m = 0;
    while (getline(file, line)) {
        std::stringstream stream(line);
        std::string col;
        m = 0;
        while (getline(stream, col, ',')) {
            std::istringstream num_stream(col);
            num_stream >> value;
            text.push_back(value);
            m++;
        }
        n++;
    }
    file.close();

    return EXIT_SUCCESS;
}