#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <cstdlib>

#include "hip/hip_runtime.h"
#include ""
#include "../include/matrix.h"
#include "../include/timer.h"
#include "../include/wrappers.cuh"

#define debug 1

int main(int argc, char *argv[]) {
    // Filename given as input from console.
    if (argc != 3) {
        std::cerr << "Syntax: ./<executable> <filename> <blocksize per dimension> \n"
                  << " *Note: Some kernels use the same blocksize for dimension X and Y \n"
                  << " *Note: The resulting blocksize is (e.g for input n) n x n x 1 for each dim (x, y, z) \n \n"
                  << "**Note: Blocksize for each dimension X & Y much larger than 32 \n"
                  << "**Note: is not supported for cards with comp.capability < 6.0" << std::endl;
        return EXIT_FAILURE;
    }
    // Convert the text argument to integer.
    int block_num = 0;
    std::istringstream block_stream(argv[2]);
    block_stream >> block_num;
    
    // Bounds Checking for the blocksize.
    if (block_num < 0) {
        std::cerr << " *Error: blocksize for each dimension should be a positive number" << std::endl;
        return EXIT_FAILURE;
    }
    else if (block_num < 16) {
        std::cerr << " *Error: the number of threads for each dimension exceeds 65535 \n (e.g 1.000.000 / 15 > 65335) \n"
                  << " *Note:  use a blocksize for each dimension >= 16" << std::endl;
        return EXIT_FAILURE;
    }
    else if (block_num > 32) {
        std::cerr << " *Error: the number of blocks for both dimensions exceeds 1024 \n (e.g 33x33 = 1089) \n"
                  <<  " *Note:  use a blocksize for each dimension <= 32" << std::endl;
        return EXIT_FAILURE;
    }

    std::ifstream file(argv[1]);
    // Checking if file stream has succesfully opened.
    if (!file.is_open()) {
        std::cerr << "File doesn't exist, or failed to open!" << std::endl;
        return EXIT_FAILURE;
    }

    // Load the float values on a dynamically expanding vector.
    std::vector<double> text;
    std::string line;
    double value = 0;
    int n = 0, m = 0;
    while (getline(file, line)) {
        std::stringstream stream(line);
        std::string col;
        m = 0;
        while (getline(stream, col, ',')) {
            std::istringstream num_stream(col);
            num_stream >> value;
            text.push_back(value);
            m++;
        }
        n++;
    }
    file.close();

    matrix M(n, m);
    matrix w(n, 1);
    matrix x(m, 1);
    matrix x_prev(m, 1);

    matrix dev_M(n, m);
    matrix dev_w(n, 1);
    matrix dev_x(m, 1);
    matrix dev_x_prev(m, 1);
    matrix dev_s(n, 1);

    double epsilon = pow(10, -6);
    double delta_norm, norm;
    double delta_norm_square, norm_square;
    float msec = 0.f;
    float avg_Msec = 0.f;
    double sec = 0.0;
    double avg_sec = 0.0;
    double total_msec = 0.0;
    int count = 0;

    M.data = (double *)malloc(M.size * sizeof(double));
    w.data = (double *)malloc(w.size * sizeof(double));
    x.data = (double *)malloc(x.size * sizeof(double));
    x_prev.data = (double *)malloc(x_prev.size * sizeof(double));

    // Load M on the Host system memory.
    for (int row = 0; row < M.rows; ++row) {
        for (int col = 0; col < M.cols; ++col) {
            M.data[idx(row, M.cols, col)] = text[idx(row, M.cols, col)];
        }
    }
    text.clear(); // Empty the vector.

    // Initialize w values to 0
    for (int index = 0; index < n; ++index) {
        w.data[index] = 0.0;
    }

    hipError_t cudaStatus; // Return value of each cuda call.

    // Choose which GPU Device to execute on.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed!";

        // Free all memory both in Device and CPU.
        hipFree(dev_M.data);
        hipFree(dev_w.data);
        hipFree(dev_x.data);
        hipFree(dev_x_prev.data);

        free(M.data);
        free(w.data);
        free(x.data);
        free(x_prev.data);

        return EXIT_FAILURE;
    }

    hipMalloc(&dev_M.data, dev_M.size * sizeof(double));
    hipMalloc(&dev_w.data, dev_w.size * sizeof(double));
    hipMalloc(&dev_x.data, dev_x.size * sizeof(double));
    hipMalloc(&dev_x_prev.data, dev_x_prev.size * sizeof(double));
    hipMalloc(&dev_s.data, dev_s.size * sizeof(double));

    initialize_vector(dev_x, 0, block_num);
    initialize_vector(dev_x_prev, 1, block_num);

    cudaStatus = hipMemcpy(dev_M.data, M.data, dev_M.size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed! 1";

        // Free all memory both in Device and CPU.
        hipFree(dev_M.data);
        hipFree(dev_w.data);
        hipFree(dev_x.data);
        hipFree(dev_x_prev.data);

        free(M.data);
        free(w.data);
        free(x.data);
        free(x_prev.data);

        return EXIT_FAILURE;
    }
    initialize_w(dev_M, dev_w, block_num);
    cudaStatus = hipMemcpy(w.data, dev_w.data, dev_w.size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed! 2";

        // Free all memory both in Device and CPU.
        hipFree(dev_M.data);
        hipFree(dev_w.data);
        hipFree(dev_x.data);
        hipFree(dev_x_prev.data);

        free(M.data);
        free(w.data);
        free(x.data);
        free(x_prev.data);

        return EXIT_FAILURE;
    }
    delta_norm = 0;
    norm = 0;
    
    do {
        ++count; // Count the amount of iterations.
        msec = Multiply(dev_M, dev_w, dev_x, dev_x_prev, dev_s, block_num); // Multiply and return the elapsed time.

#if debug
        std::cout << "Multiply time elapsed: " << msec << " msec\n";
#endif      
        avg_Msec += msec;
        // Transfer Xk, Xk-1 back to cpu in order to calculate the norm.
        hipMemcpy(x.data, dev_x.data, dev_x.size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(x_prev.data, dev_x_prev.data, dev_x_prev.size * sizeof(double), hipMemcpyDeviceToHost);

        // Start Measuring - Calculate norm 
        start_timer
            // Calculate norm ^ 2 = sum(Xk[i] ^ 2) and norm
            norm_square = 0;
        for (int row = 0; row < x.rows; ++row) {
            norm_square += std::pow(x.data[row], 2);
        }
        norm = std::sqrt(norm_square);

        // Normalize Xk.
        for (int row = 0; row < x.rows; ++row) {
            x.data[row] /= norm;
        }

        // Calculate ||Xk - Xk-1|| ^ 2 and ||Xk - Xk-1||.
        delta_norm_square = 0;
        for (int row = 0; row < x_prev.size && x.size; ++row) {
            delta_norm_square += std::pow((x.data[row] - x_prev.data[row]), 2);
        }
        delta_norm = std::sqrt(delta_norm_square);
        // Stop Measuring - Norm calculated.
        stop_timer

            if (delta_norm > epsilon) {
                // Calculate the average time in seconds for norm calculation.
#ifdef _WIN32
                std::chrono::duration<double> elapsed_seconds = stop - start;
                sec = elapsed_seconds.count();
#else
                sec = (stop.tv_sec - start.tv_sec) + ((stop.tv_usec - start.tv_usec) / 1000000.0);
#endif
                avg_sec += sec;
                total_msec += (sec * 1000) + msec;
#if debug
                std::cout << "delta_norm " << delta_norm << "\n"
                          << "Elapsed seconds for Norm: " << sec << " sec \n"
                          << std::endl;
#endif
                // Transfer the current Xk (Host) to the new Xk-1 (Device)!!
                hipMemcpy(dev_x_prev.data, x.data, dev_x.size * sizeof(double), hipMemcpyHostToDevice);

                if (cudaStatus != hipSuccess) {
                    std::cerr << "hipMemcpy failed to swap xk and xk - 1!";

                    // Free all memory both in Device and CPU.
                    hipFree(dev_M.data);
                    hipFree(dev_w.data);
                    hipFree(dev_x.data);
                    hipFree(dev_x_prev.data);

                    free(M.data);
                    free(w.data);
                    free(x.data);
                    free(x_prev.data);

                    return EXIT_FAILURE;
                }

                // Re - initialize the new Xk to 0.
                initialize_vector(dev_x, 0, block_num);
            }
    } while (delta_norm > epsilon);

    // Print the final average times.
    avg_sec /= count;
    avg_Msec /= count;
#if debug
    std::cout << std::endl
              << "Average Multiplication time: " << avg_Msec << " msec \n"
              << "Average Norm Calculation time: " << avg_sec * 1000 << " msec \n"
              << "Average Iteration time: " << avg_Msec + (avg_sec * 1000) << " msec \n\n"
              << "Total Number of iterations: " << count << "\n"
              << "Total Time of all iterations: " << total_msec / 1000 << " sec" << std::endl;
#endif
    // Open output stream for file write only.
    std::ofstream result("results.csv");

    for (int row = 0; row < x.rows - 1; row++) {
        result << x.data[row] << ",";
    }
    // Unroll the last iteration of the loop above.
    result << x.data[x.rows - 1] << std::endl;
    // Close the file.
    result.close();
    
    hipFree(dev_M.data);
    hipFree(dev_w.data);
    hipFree(dev_x.data);
    hipFree(dev_x_prev.data);
    hipFree(dev_s.data);

    free(M.data);
    free(w.data);
    free(x.data);
    free(x_prev.data);
    return EXIT_SUCCESS;
}