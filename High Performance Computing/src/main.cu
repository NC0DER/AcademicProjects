#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <cstdlib>

#include "hip/hip_runtime.h"
#include ""
#include "../include/matrix.h"
#include "../include/wrappers.cuh"

#define debug 1

int main(int argc, char *argv[]) {
    // Filename given as input from console.
    if (argc != 3) {
        std::cerr << "Syntax: ./<executable> <filename> <blocksize per dimension> \n"
                  << " *Note: Some kernels use the same blocksize for dimension X and Y \n"
                  << " *Note: The resulting blocksize is (e.g for input n) n x n x 1 for each dim (x, y, z) \n \n"
                  << "**Note: Blocksize for each dimension X & Y much larger than 32 \n"
                  << "**Note: is not supported for cards with comp.capability < 6.0" << std::endl;
        return EXIT_FAILURE;
    }
    // Convert the text argument to integer.
    int block_num = 0;
    std::istringstream block_stream(argv[2]);
    block_stream >> block_num;
    
    return EXIT_SUCCESS;
}