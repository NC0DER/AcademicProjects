#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <cstdlib>

#include "hip/hip_runtime.h"
#include ""
#include "../include/matrix.h"
#include "../include/wrappers.cuh"

#define debug 1

int main(int argc, char *argv[]) {
    // Filename given as input from console.
    if (argc != 3) {
        std::cerr << "Syntax: ./<executable> <filename> <blocksize per dimension> \n"
                  << " *Note: Some kernels use the same blocksize for dimension X and Y \n"
                  << " *Note: The resulting blocksize is (e.g for input n) n x n x 1 for each dim (x, y, z) \n \n"
                  << "**Note: Blocksize for each dimension X & Y much larger than 32 \n"
                  << "**Note: is not supported for cards with comp.capability < 6.0" << std::endl;
        return EXIT_FAILURE;
    }
    // Convert the text argument to integer.
    int block_num = 0;
    std::istringstream block_stream(argv[2]);
    block_stream >> block_num;
    
    std::ifstream file(argv[1]);
    // Checking if file stream has succesfully opened.
    if (!file.is_open()) {
        std::cerr << "File doesn't exist, or failed to open!" << std::endl;
        return EXIT_FAILURE;
    }

    // Load the float values on a dynamically expanding vector.
    std::vector<double> text;
    std::string line;
    double value = 0;
    int n = 0, m = 0;
    while (getline(file, line)) {
        std::stringstream stream(line);
        std::string col;
        m = 0;
        while (getline(stream, col, ',')) {
            std::istringstream num_stream(col);
            num_stream >> value;
            text.push_back(value);
            m++;
        }
        n++;
    }
    file.close();

    return EXIT_SUCCESS;
}